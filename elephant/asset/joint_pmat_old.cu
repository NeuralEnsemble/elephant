#include "hip/hip_runtime.h"
/**
 * CUDA implementation of ASSET.joint_probability_matrix function (refer to
 * Python documentation).
 */

#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <math.h>
#include <assert.h>
#include <float.h>
#include <limits.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define L {{L}}
#define N {{N}}
#define D {{D}}

#if D > N
#error "D must be less or equal N"
#endif

#define min_macros(a,b)   ((a) < (b) ? (a) : (b))

#define ASSET_DEBUG       {{ASSET_DEBUG}}
#define ULL               unsigned long long


/**
 * The maximum number of threads per block.
 * This number must be in range [1, 1024].
 * The effective number of threads will be set dynamically
 * at runtime to match the tile (width L) of a block.
 */
#define N_THREADS         {{N_THREADS}}

/**
 * To reduce branch divergence in 'next_sequence_sorted' function
 * within a warp (threads in a warp take different branches),
 * each thread runs CWR_LOOPS of 'combinations_with_replacement'.
 */
#define CWR_LOOPS         {{CWR_LOOPS}}

#define L_BLOCK_SUPREMUM  min_macros(N_THREADS, L)

typedef {{precision}} asset_float;

__constant__ asset_float log_factorial[N + 1];
__constant__ asset_float logK;
__constant__ ULL ITERATIONS_TODO;
__constant__ ULL L_BLOCK;
__constant__ ULL L_NUM_BLOCKS;
__constant__ ULL iteration_table[D][N];  /* Maps the iteration ID to the entries
                                            of a sequence_sorted array */

/**
 * Compute capabilities lower than 6.0 don't have hardware support for
 * double-precision atomicAdd. This software implementation is taken from
 * https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html
 */
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 600
__device__ double atomicAdd(double* address, double val)
{
    ULL* address_as_ull = (ULL*)address;
    ULL old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}
#endif

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


/**
 * Builds the next sequence_sorted, given the absolute iteration ID.
 * The time complexity is O(N+D), not O(N*D).
 *
 * @param sequence_sorted the output sequence_sorted array of size D
 * @param iteration       the global iteration ID
 */
__device__ void next_sequence_sorted(int *sequence_sorted, ULL iteration) {
    int row, element = N - 1;
    for (row = D - 1; row >= 0; row--) {
        while (element > row && iteration < iteration_table[row][element]) {
            element--;
        }
        iteration -= iteration_table[row][element];
        sequence_sorted[D - 1 - row] = element + 1;
    }
}


/**
 * Set 'sequence_sorted' to the next valid sequence of indices in-place.
 */
__device__ void combinations_with_replacement(int *sequence_sorted) {
    int increment_id = D - 1;
    while (increment_id > 0 && sequence_sorted[increment_id - 1] == sequence_sorted[increment_id]) {
      sequence_sorted[increment_id] = D - increment_id;
      increment_id--;
    }
    sequence_sorted[increment_id]++;
}


/**
 * CUDA kernel that computes P_total - the joint survival probabilities matrix.
 *
 * @param P_out           P_total output array of size L
 * @param log_du_device   input log_du flattened matrix of size L*(D+1)
 */
__global__ void jsf_uniform_orderstat_3d_kernel(asset_float *P_out, const float *log_du_device) {
    unsigned int i;
    ULL row;

    // the row shift of log_du and P_total in the number of elements, between 0 and L
    const ULL l_shift = (blockIdx.x % L_NUM_BLOCKS) * L_BLOCK;

    // account for the last block width that can be less than L_BLOCK
    const ULL block_width = (L - l_shift < L_BLOCK) ? (L - l_shift) : L_BLOCK;

    extern __shared__ float shared_mem[];
    asset_float *P_total = (asset_float*) shared_mem;  // L_BLOCK floats
    float *log_du = (float*)&P_total[L_BLOCK];       // L_BLOCK * (D + 1) floats

    for (row = threadIdx.x; row < block_width; row += blockDim.x) {
        P_total[row] = 0;
        for (i = 0; i <= D; i++) {
            log_du[row * (D + 1) + i] = log_du_device[(row + l_shift) * (D + 1) + i];
        }
    }

    __syncthreads();

    int di[D + 1];
    int sequence_sorted[D];
    asset_float P_thread[L_BLOCK_SUPREMUM];
    for (row = 0; row < block_width; row++) {
        P_thread[row] = 0;
    }

    const ULL burnout = (blockIdx.x / L_NUM_BLOCKS) * blockDim.x * CWR_LOOPS + threadIdx.x * CWR_LOOPS;
    const ULL stride = (gridDim.x / L_NUM_BLOCKS) * blockDim.x * CWR_LOOPS;

    ULL iteration, cwr_loop;
    for (iteration = burnout; iteration < ITERATIONS_TODO; iteration += stride) {
        next_sequence_sorted(sequence_sorted, iteration);

        for (cwr_loop = 0; (cwr_loop < CWR_LOOPS) && (sequence_sorted[0] != N + 1); cwr_loop++) {
            int prev = N;
            for (i = 0; i < D; i++) {
                di[i] = prev - sequence_sorted[i];
                prev = sequence_sorted[i];
            }
            di[D] = sequence_sorted[D - 1];

            asset_float sum_log_di_factorial = 0.f;
            for (i = 0; i <= D; i++) {
                sum_log_di_factorial += log_factorial[di[i]];
            }

            asset_float colsum;
            const asset_float colsum_base = logK - sum_log_di_factorial;
            const float *log_du_row = log_du;
            for (row = 0; row < block_width; row++) {
                colsum = colsum_base;
                for (i = 0; i <= D; i++) {
                    if (di[i] != 0) {
                        colsum += di[i] * log_du_row[i];
                    }
                }
                P_thread[row] += exp(colsum);
                log_du_row += D + 1;
            }

            combinations_with_replacement(sequence_sorted);
        }
    }

    for (row = threadIdx.x; row < block_width + threadIdx.x; row++) {
        // Reduce atomicAdd conflicts by adding threadIdx.x to each row
        atomicAdd(P_total + row % block_width, P_thread[row % block_width]);
    }

    __syncthreads();

    for (row = threadIdx.x; row < block_width; row += blockDim.x) {
        atomicAdd(P_out + row + l_shift, P_total[row]);
    }
}


/**
 * Creates a flattened matrix (D-1)*N that will be used
 * to map the iteration ID to a sequence_sorted array.
 */
ULL create_iteration_table() {
    ULL *m = (ULL*) calloc(D * N, sizeof(ULL));
    unsigned int row, col;
    for (col = 0; col < N; col++) {
        m[col] = col;
    }
    for (row = 1; row < D; row++) {
        ULL sum = 0;
        for (col = row + 1; col < N; col++) {
            sum += m[(row - 1) * N + col];
            m[row * N + col] = sum;
        }
    }

    ULL it_todo = 1;
    double it_todo_double = 1.0;
    for (row = 0; row < D; row++) {
        it_todo += m[row * N + N-1];
        it_todo_double += m[row * N + N-1];
    }

    // check for the integer overflow;
    // values greater than ULONG_MAX are not supported by CUDA
    assert(it_todo_double <= ULONG_MAX);

    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(iteration_table), m, sizeof(ULL) * D * N) );

    gpuErrchk( hipMemcpyToSymbol((const void*) &ITERATIONS_TODO, (const void*) &it_todo, sizeof(ULL)) );

    free(m);

    return it_todo;
}


// For debugging purposes only
void print_constants() {
    int i, col;
    printf(">>> iteration_table\n");
    ULL iteration_table_host[D * N];
    hipMemcpyFromSymbol(iteration_table_host, HIP_SYMBOL(iteration_table), sizeof(ULL) * D * N);
    int row;
    for (row = 0; row < D; row++) {
        for (col = 0; col < N; col++) {
            printf("%10llu ", iteration_table_host[row * N + col]);
        }
        printf("\n");
    }
    printf("\n");

    ULL it_todo_host;
    hipMemcpyFromSymbol((void*)&it_todo_host, (const void*)&ITERATIONS_TODO, sizeof(ULL));
    printf(">>> ITERATIONS_TODO = %llu\n", it_todo_host);

    ULL l_block;
    hipMemcpyFromSymbol((void*)&l_block, (const void*)&L_BLOCK, sizeof(ULL));
    printf(">>> L_BLOCK = %llu\n", l_block);

    ULL l_num_blocks;
    hipMemcpyFromSymbol((void*)&l_num_blocks, (const void*)&L_NUM_BLOCKS, sizeof(ULL));
    printf(">>> L_NUM_BLOCKS = %llu\n", l_num_blocks);

    asset_float logK_host;
    hipMemcpyFromSymbol((void*)&logK_host, (const void*)&logK, sizeof(asset_float));
    printf(">>> logK = %f\n\n", logK_host);

    asset_float log_factorial_host[N + 1];
    hipMemcpyFromSymbol(log_factorial_host, HIP_SYMBOL(log_factorial), sizeof(asset_float) * (N+1));
    printf(">>> log_factorial\n");
    for (i = 0; i <= N; i++) {
        printf("%f ", log_factorial_host[i]);
    }
    printf("\n\n");
}


/**
 * ASSET jsf_uniform_orderstat_3d host function to calculate P_total.
 * The result of a calculation is saved in P_total_host array.
 *
 * @param P_total_host a pointer to P_total array to be calculated
 * @param log_du_host  input flattened L*(D+1) matrix of log_du values
 */
void jsf_uniform_orderstat_3d(asset_float *P_total_host, FILE *log_du_file) {
    float *log_du_device;
    gpuErrchk( hipMalloc((void**)&log_du_device, sizeof(float) * L * (D + 1)) );

    float *log_du_host;

#if L * (D + 1) < 100000000LLU
    // For arrays of size <100 Mb, allocate host memory for log_du
    log_du_host = (float*) malloc(sizeof(float) * L * (D + 1));
    fread(log_du_host, sizeof(float), L * (D + 1), log_du_file);
    gpuErrchk( hipMemcpyAsync(log_du_device, log_du_host, sizeof(float) * L * (D + 1), hipMemcpyHostToDevice) );
#else
    // Use P_total buffer to read log_du and copy batches to a GPU card
    log_du_host = (float*) P_total_host;
    ULL col;
    for (col = 0; col <= D; col++) {
        fread(log_du_host, sizeof(float), L, log_du_file);
        // Wait till the copy finishes before filling the buffer with a next chunk.
        gpuErrchk( hipMemcpy(log_du_device + col * L, log_du_host, sizeof(float) * L, hipMemcpyHostToDevice) );
    }
#endif

    fclose(log_du_file);

    asset_float *P_total_device;

    // Initialize P_total_device with zeros.
    // Note that values other than 0x00 or 0xFF (NaN) won't work
    // with hipMemset when the data type is float or double.
    gpuErrchk( hipMalloc((void**)&P_total_device, sizeof(asset_float) * L) );
    gpuErrchk( hipMemsetAsync(P_total_device, 0, sizeof(asset_float) * L) );

    ULL it_todo = create_iteration_table();

    asset_float logK_host = 0.f;
    asset_float log_factorial_host[N + 1] = {0.f};

    int i;
    for (i = 1; i <= N; i++) {
        logK_host += log((asset_float) i);
        log_factorial_host[i] = logK_host;
    }

    gpuErrchk( hipMemcpyToSymbol((const void*) &logK, (const void*) &logK_host, sizeof(asset_float)) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(log_factorial), log_factorial_host, sizeof(asset_float) * (N + 1)) );

    hipDeviceProp_t device_prop;
    gpuErrchk( hipGetDeviceProperties(&device_prop, 0) );
    const ULL max_l_block = device_prop.sharedMemPerBlock / (sizeof(asset_float) * (D + 2));

    /**
     * It's not necessary to match N_THREADS with the final L_BLOCK. Alternatively,
     * the desired L_BLOCK can be another parameter specified by the user. But
     * the optimal L_BLOCK on average matches N_THREADS, therefore, to avoid
     * the user thinking too much, we take care of the headache by setting
     * L_BLOCK = N_THREADS.
     */
    unsigned int n_threads = (unsigned int) min_macros(N_THREADS, min_macros(max_l_block, device_prop.maxThreadsPerBlock));
    if (n_threads > device_prop.warpSize) {
        // It's more efficient to make the number of threads
        // a multiple of the warp size (32).
        n_threads -= n_threads % device_prop.warpSize;
    }
    const ULL l_block = min_macros(n_threads, L);
    gpuErrchk( hipMemcpyToSymbol((const void*) &L_BLOCK, (const void*) &l_block, sizeof(ULL)) );

    const ULL l_num_blocks = (ULL) ceil(L * 1.f / l_block);
    gpuErrchk( hipMemcpyToSymbol((const void*) &L_NUM_BLOCKS, (const void*) &l_num_blocks, sizeof(ULL)) );

    ULL grid_size = (ULL) ceil(it_todo * 1.f / (n_threads * CWR_LOOPS));
    grid_size = min_macros(grid_size, device_prop.maxGridSize[0]);
    if (grid_size > l_num_blocks) {
        // make grid_size divisible by l_num_blocks
        grid_size -= grid_size % l_num_blocks;
    } else {
        // grid_size must be at least l_num_blocks
        grid_size = l_num_blocks;
    }

    printf(">>> it_todo=%llu, grid_size=%llu, L_BLOCK=%llu, N_THREADS=%u\n\n", it_todo, grid_size, l_block, n_threads);

    // Wait for asynchronous memory copies to finish.
    gpuErrchk( hipDeviceSynchronize() );

    if (log_du_host != (float*) P_total_host) {
        // the memory has been allocated
        free(log_du_host);
    }

#if ASSET_DEBUG
    print_constants();
#endif

    // Executing the kernel
    const ULL shared_mem_used = sizeof(asset_float) * l_block + sizeof(float) * l_block * (D + 1);
    jsf_uniform_orderstat_3d_kernel<<<grid_size, n_threads, shared_mem_used>>>(P_total_device, log_du_device);

    // Check for invalid launch argument.
    gpuErrchk( hipPeekAtLastError() );

    // Transfer data back to host memory.
    // If the exit code is non-zero, the kernel failed to complete the task.
    hipError_t cuda_completed_status = hipMemcpy(P_total_host, P_total_device, sizeof(asset_float) * L, hipMemcpyDeviceToHost);

    hipFree(P_total_device);
    hipFree(log_du_device);

    gpuErrchk( cuda_completed_status );
}


int main(int argc, char* argv[]) {
    // compile command: nvcc -o asset.o asset.cu
    // (run after you fill the template keys L, N, D, etc.)
    if (argc != 3) {
        fprintf(stderr, "Usage: ./asset.o /path/to/log_du.dat /path/to/P_total_output.dat\n");
        return 1;
    }
    char *log_du_path = argv[1];
    char *P_total_path = argv[2];

    FILE *log_du_file = fopen(log_du_path, "rb");

    if (log_du_file == NULL) {
        fprintf(stderr, "File '%s' not found\n", log_du_path);
        return 1;
    }

    asset_float *P_total = (asset_float*) malloc(sizeof(asset_float) * L);

    jsf_uniform_orderstat_3d(P_total, log_du_file);

    FILE *P_total_file = fopen(P_total_path, "wb");
    if (P_total_file == NULL) {
        free(P_total);
        fprintf(stderr, "Could not open '%s' for writing.\n", P_total_path);
        return 1;
    }
    fwrite(P_total, sizeof(asset_float), L, P_total_file);
    fclose(P_total_file);

    free(P_total);

    return 0;
}
